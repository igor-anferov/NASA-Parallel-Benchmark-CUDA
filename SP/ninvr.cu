#include "hip/hip_runtime.h"
//-------------------------------------------------------------------------//
//                                                                         //
//  This benchmark is a serial C version of the NPB SP code. This C        //
//  version is developed by the Center for Manycore Programming at Seoul   //
//  National University and derived from the serial Fortran versions in    //
//  "NPB3.3-SER" developed by NAS.                                         //
//                                                                         //
//  Permission to use, copy, distribute and modify this software for any   //
//  purpose with or without fee is hereby granted. This software is        //
//  provided "as is" without express or implied warranty.                  //
//                                                                         //
//  Information on NPB 3.3, including the technical report, the original   //
//  specifications, source code, results and information on how to submit  //
//  new results, is available at:                                          //
//                                                                         //
//           http://www.nas.nasa.gov/Software/NPB/                         //
//                                                                         //
//  Send comments or suggestions for this C version to cmp@aces.snu.ac.kr  //
//                                                                         //
//          Center for Manycore Programming                                //
//          School of Computer Science and Engineering                     //
//          Seoul National University                                      //
//          Seoul 151-744, Korea                                           //
//                                                                         //
//          E-mail:  cmp@aces.snu.ac.kr                                    //
//                                                                         //
//-------------------------------------------------------------------------//

//-------------------------------------------------------------------------//
// Authors: Sangmin Seo, Jungwon Kim, Jun Lee, Jeongho Nah, Gangwon Jo,    //
//          and Jaejin Lee                                                 //
//-------------------------------------------------------------------------//

#include "header.h"

//---------------------------------------------------------------------
// block-diagonal matrix-vector multiplication              
//---------------------------------------------------------------------
__global__ void ninvr_kernel(
    int nx2, int ny2, int nz2,
    double (*rhs)[KMAX][JMAXP+1][IMAXP+1]
) {
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  int j = blockDim.y * blockIdx.y + threadIdx.y;
  int k = blockDim.z * blockIdx.z + threadIdx.z;
  double r1, r2, r3, r4, r5, t1, t2;

  if (k >= 1 && k <= nz2) {
    if (j >= 1 && j <= ny2) {
      if (i >= 1 && i <= nx2) {
        r1 = rhs[0][k][j][i];
        r2 = rhs[1][k][j][i];
        r3 = rhs[2][k][j][i];
        r4 = rhs[3][k][j][i];
        r5 = rhs[4][k][j][i];

        t1 = bt * r3;
        t2 = 0.5 * ( r4 + r5 );

        rhs[0][k][j][i] = -r2;
        rhs[1][k][j][i] =  r1;
        rhs[2][k][j][i] = bt * ( r4 - r5 );
        rhs[3][k][j][i] = -t1 + t2;
        rhs[4][k][j][i] =  t1 + t2;
      }
    }
  }
}

void ninvr() {
  if (timeron) timer_start(t_ninvr);
  ninvr_kernel <<< gridDim_, blockDim_ >>> (
    nx2, ny2, nz2, device_rhs
  );
//  assert(hipSuccess == hipDeviceSynchronize());
  if (timeron) timer_stop(t_ninvr);
}
