#include "hip/hip_runtime.h"
//-------------------------------------------------------------------------//
//                                                                         //
//  This benchmark is a serial C version of the NPB SP code. This C        //
//  version is developed by the Center for Manycore Programming at Seoul   //
//  National University and derived from the serial Fortran versions in    //
//  "NPB3.3-SER" developed by NAS.                                         //
//                                                                         //
//  Permission to use, copy, distribute and modify this software for any   //
//  purpose with or without fee is hereby granted. This software is        //
//  provided "as is" without express or implied warranty.                  //
//                                                                         //
//  Information on NPB 3.3, including the technical report, the original   //
//  specifications, source code, results and information on how to submit  //
//  new results, is available at:                                          //
//                                                                         //
//           http://www.nas.nasa.gov/Software/NPB/                         //
//                                                                         //
//  Send comments or suggestions for this C version to cmp@aces.snu.ac.kr  //
//                                                                         //
//          Center for Manycore Programming                                //
//          School of Computer Science and Engineering                     //
//          Seoul National University                                      //
//          Seoul 151-744, Korea                                           //
//                                                                         //
//          E-mail:  cmp@aces.snu.ac.kr                                    //
//                                                                         //
//-------------------------------------------------------------------------//

//-------------------------------------------------------------------------//
// Authors: Sangmin Seo, Jungwon Kim, Jun Lee, Jeongho Nah, Gangwon Jo,    //
//          and Jaejin Lee                                                 //
//-------------------------------------------------------------------------//

#include <math.h>
#include <assert.h>
#include "header.h"

__global__ void compute_rhs_intro(
    dim3 gridOffset,
    int* grid_points,
    double (*u      )/*[KMAX]*/[JMAXP+1][IMAXP+1][5],
    double (*us     )/*[KMAX]*/[JMAXP+1][IMAXP+1],
    double (*vs     )/*[KMAX]*/[JMAXP+1][IMAXP+1],
    double (*ws     )/*[KMAX]*/[JMAXP+1][IMAXP+1],
    double (*qs     )/*[KMAX]*/[JMAXP+1][IMAXP+1],
    double (*rho_i  )/*[KMAX]*/[JMAXP+1][IMAXP+1],
    double (*speed  )/*[KMAX]*/[JMAXP+1][IMAXP+1],
    double (*square )/*[KMAX]*/[JMAXP+1][IMAXP+1],
    double (*rhs    )/*[KMAX]*/[JMAXP+1][IMAXP+1][5],
    double (*forcing)/*[KMAX]*/[JMAXP+1][IMAXP+1][5]
) {
  int i = blockDim.x * blockIdx.x + threadIdx.x + gridOffset.x;
  int j = blockDim.y * blockIdx.y + threadIdx.y + gridOffset.y;
  int k = blockDim.z * blockIdx.z + threadIdx.z + gridOffset.z;

  int m;
  double rho_inv, aux;
  if (k >= 0 && k <= grid_points[2]-1) {
    if (j >= 0 && j <= grid_points[1]-1) {
      if (i >= 0 && i <= grid_points[0]-1) {
        rho_inv = 1.0/u[k][j][i][0];
        rho_i[k][j][i] = rho_inv;
        us[k][j][i] = u[k][j][i][1] * rho_inv;
        vs[k][j][i] = u[k][j][i][2] * rho_inv;
        ws[k][j][i] = u[k][j][i][3] * rho_inv;
        square[k][j][i] = 0.5* (
            u[k][j][i][1]*u[k][j][i][1] + 
            u[k][j][i][2]*u[k][j][i][2] +
            u[k][j][i][3]*u[k][j][i][3] ) * rho_inv;
        qs[k][j][i] = square[k][j][i] * rho_inv;
        //-------------------------------------------------------------------
        // (don't need speed and ainx until the lhs computation)
        //-------------------------------------------------------------------
        aux = c1c2*rho_inv* (u[k][j][i][4] - square[k][j][i]);
        speed[k][j][i] = sqrt(aux);
      }
    }
  }

  //---------------------------------------------------------------------
  // copy the exact forcing term to the right hand side;  because 
  // this forcing term is known, we can store it on the whole grid
  // including the boundary                   
  //---------------------------------------------------------------------
  if (k >= 0 && k <= grid_points[2]-1) {
    if (j >= 0 && j <= grid_points[1]-1) {
      if (i >= 0 && i <= grid_points[0]-1) {
#pragma unroll
        for (m = 0; m < 5; m++) {
          rhs[k][j][i][m] = forcing[k][j][i][m];
        }
      }
    }
  }
}

__global__ void compute_rhs_xi(
    dim3 gridOffset,
    int nx2, int ny2, int nz2,
    double (*u      )/*[KMAX]*/[JMAXP+1][IMAXP+1][5],
    double (*us     )/*[KMAX]*/[JMAXP+1][IMAXP+1],
    double (*vs     )/*[KMAX]*/[JMAXP+1][IMAXP+1],
    double (*ws     )/*[KMAX]*/[JMAXP+1][IMAXP+1],
    double (*qs     )/*[KMAX]*/[JMAXP+1][IMAXP+1],
    double (*rho_i  )/*[KMAX]*/[JMAXP+1][IMAXP+1],
    double (*square )/*[KMAX]*/[JMAXP+1][IMAXP+1],
    double (*rhs    )/*[KMAX]*/[JMAXP+1][IMAXP+1][5],
    double dx1tx1, double dx2tx1, double dx3tx1, double dx4tx1, double dx5tx1, double tx2,
    double xxcon2, double xxcon3, double xxcon4, double xxcon5
) {
  int i = blockDim.x * blockIdx.x + threadIdx.x + gridOffset.x;
  int j = blockDim.y * blockIdx.y + threadIdx.y + gridOffset.y;
  int k = blockDim.z * blockIdx.z + threadIdx.z + gridOffset.z;

  int m;
  double uijk, up1, um1;
  if (k >= 1 && k <= nz2) {
    if (j >= 1 && j <= ny2) {
      if (i >= 1 && i <= nx2) {
        uijk = us[k][j][i];
        up1  = us[k][j][i+1];
        um1  = us[k][j][i-1];

        rhs[k][j][i][0] = rhs[k][j][i][0] + dx1tx1 * 
          (u[k][j][i+1][0] - 2.0*u[k][j][i][0] + u[k][j][i-1][0]) -
          tx2 * (u[k][j][i+1][1] - u[k][j][i-1][1]);

        rhs[k][j][i][1] = rhs[k][j][i][1] + dx2tx1 * 
          (u[k][j][i+1][1] - 2.0*u[k][j][i][1] + u[k][j][i-1][1]) +
          xxcon2*con43 * (up1 - 2.0*uijk + um1) -
          tx2 * (u[k][j][i+1][1]*up1 - u[k][j][i-1][1]*um1 +
                (u[k][j][i+1][4] - square[k][j][i+1] -
                 u[k][j][i-1][4] + square[k][j][i-1]) * c2);

        rhs[k][j][i][2] = rhs[k][j][i][2] + dx3tx1 * 
          (u[k][j][i+1][2] - 2.0*u[k][j][i][2] + u[k][j][i-1][2]) +
          xxcon2 * (vs[k][j][i+1] - 2.0*vs[k][j][i] + vs[k][j][i-1]) -
          tx2 * (u[k][j][i+1][2]*up1 - u[k][j][i-1][2]*um1);

        rhs[k][j][i][3] = rhs[k][j][i][3] + dx4tx1 * 
          (u[k][j][i+1][3] - 2.0*u[k][j][i][3] + u[k][j][i-1][3]) +
          xxcon2 * (ws[k][j][i+1] - 2.0*ws[k][j][i] + ws[k][j][i-1]) -
          tx2 * (u[k][j][i+1][3]*up1 - u[k][j][i-1][3]*um1);

        rhs[k][j][i][4] = rhs[k][j][i][4] + dx5tx1 * 
          (u[k][j][i+1][4] - 2.0*u[k][j][i][4] + u[k][j][i-1][4]) +
          xxcon3 * (qs[k][j][i+1] - 2.0*qs[k][j][i] + qs[k][j][i-1]) +
          xxcon4 * (up1*up1 -       2.0*uijk*uijk + um1*um1) +
          xxcon5 * (u[k][j][i+1][4]*rho_i[k][j][i+1] - 
                2.0*u[k][j][i][4]*rho_i[k][j][i] +
                    u[k][j][i-1][4]*rho_i[k][j][i-1]) -
          tx2 * ( (c1*u[k][j][i+1][4] - c2*square[k][j][i+1])*up1 -
                  (c1*u[k][j][i-1][4] - c2*square[k][j][i-1])*um1 );
      }
    }

    //---------------------------------------------------------------------
    // add fourth order xi-direction dissipation               
    //---------------------------------------------------------------------
    if (j >= 1 && j <= ny2) {
      if (i == 1)
#pragma unroll
          for (m = 0; m < 5; m++) {
            rhs[k][j][i][m] = rhs[k][j][i][m]- dssp * 
              (5.0*u[k][j][i][m] - 4.0*u[k][j][i+1][m] + u[k][j][i+2][m]);
          }

      if (i == 2)
#pragma unroll
          for (m = 0; m < 5; m++) {
            rhs[k][j][i][m] = rhs[k][j][i][m] - dssp * 
              (-4.0*u[k][j][i-1][m] + 6.0*u[k][j][i][m] -
                4.0*u[k][j][i+1][m] + u[k][j][i+2][m]);
          }
    }

    if (j >= 1 && j <= ny2) {
      if (i >= 3 && i <= nx2-2) {
#pragma unroll
        for (m = 0; m < 5; m++) {
          rhs[k][j][i][m] = rhs[k][j][i][m] - dssp * 
            ( u[k][j][i-2][m] - 4.0*u[k][j][i-1][m] + 
            6.0*u[k][j][i][m] - 4.0*u[k][j][i+1][m] + 
              u[k][j][i+2][m] );
        }
      }
    }

    if (j >= 1 && j <= ny2) {
      if (i == nx2-1)
#pragma unroll
          for (m = 0; m < 5; m++) {
            rhs[k][j][i][m] = rhs[k][j][i][m] - dssp *
              ( u[k][j][i-2][m] - 4.0*u[k][j][i-1][m] + 
              6.0*u[k][j][i][m] - 4.0*u[k][j][i+1][m] );
          }

      if (i == nx2)
#pragma unroll
          for (m = 0; m < 5; m++) {
            rhs[k][j][i][m] = rhs[k][j][i][m] - dssp *
              ( u[k][j][i-2][m] - 4.0*u[k][j][i-1][m] + 5.0*u[k][j][i][m] );
          }
    }
  }
}

__global__ void compute_rhs_eta(
    dim3 gridOffset,
    int nx2, int ny2, int nz2,
    double (*u      )/*[KMAX]*/[JMAXP+1][IMAXP+1][5],
    double (*us     )/*[KMAX]*/[JMAXP+1][IMAXP+1],
    double (*vs     )/*[KMAX]*/[JMAXP+1][IMAXP+1],
    double (*ws     )/*[KMAX]*/[JMAXP+1][IMAXP+1],
    double (*qs     )/*[KMAX]*/[JMAXP+1][IMAXP+1],
    double (*rho_i  )/*[KMAX]*/[JMAXP+1][IMAXP+1],
    double (*square )/*[KMAX]*/[JMAXP+1][IMAXP+1],
    double (*rhs    )/*[KMAX]*/[JMAXP+1][IMAXP+1][5],
    double dy1ty1, double dy2ty1, double dy3ty1, double dy4ty1, double dy5ty1, double ty2,
    double yycon2, double yycon3, double yycon4, double yycon5
) {
  int i = blockDim.x * blockIdx.x + threadIdx.x + gridOffset.x;
  int j = blockDim.y * blockIdx.y + threadIdx.y + gridOffset.y;
  int k = blockDim.z * blockIdx.z + threadIdx.z + gridOffset.z;

  int m;
  double vijk, vp1, vm1;
  if (k >= 1 && k <= nz2) {
    if (j >= 1 && j <= ny2) {
      if (i >= 1 && i <= nx2) {
        vijk = vs[k][j][i];
        vp1  = vs[k][j+1][i];
        vm1  = vs[k][j-1][i];

        rhs[k][j][i][0] = rhs[k][j][i][0] + dy1ty1 * 
          (u[k][j+1][i][0] - 2.0*u[k][j][i][0] + u[k][j-1][i][0]) -
          ty2 * (u[k][j+1][i][2] - u[k][j-1][i][2]);

        rhs[k][j][i][1] = rhs[k][j][i][1] + dy2ty1 * 
          (u[k][j+1][i][1] - 2.0*u[k][j][i][1] + u[k][j-1][i][1]) +
          yycon2 * (us[k][j+1][i] - 2.0*us[k][j][i] + us[k][j-1][i]) -
          ty2 * (u[k][j+1][i][1]*vp1 - u[k][j-1][i][1]*vm1);

        rhs[k][j][i][2] = rhs[k][j][i][2] + dy3ty1 * 
          (u[k][j+1][i][2] - 2.0*u[k][j][i][2] + u[k][j-1][i][2]) +
          yycon2*con43 * (vp1 - 2.0*vijk + vm1) -
          ty2 * (u[k][j+1][i][2]*vp1 - u[k][j-1][i][2]*vm1 +
                (u[k][j+1][i][4] - square[k][j+1][i] - 
                 u[k][j-1][i][4] + square[k][j-1][i]) * c2);

        rhs[k][j][i][3] = rhs[k][j][i][3] + dy4ty1 * 
          (u[k][j+1][i][3] - 2.0*u[k][j][i][3] + u[k][j-1][i][3]) +
          yycon2 * (ws[k][j+1][i] - 2.0*ws[k][j][i] + ws[k][j-1][i]) -
          ty2 * (u[k][j+1][i][3]*vp1 - u[k][j-1][i][3]*vm1);

        rhs[k][j][i][4] = rhs[k][j][i][4] + dy5ty1 * 
          (u[k][j+1][i][4] - 2.0*u[k][j][i][4] + u[k][j-1][i][4]) +
          yycon3 * (qs[k][j+1][i] - 2.0*qs[k][j][i] + qs[k][j-1][i]) +
          yycon4 * (vp1*vp1       - 2.0*vijk*vijk + vm1*vm1) +
          yycon5 * (u[k][j+1][i][4]*rho_i[k][j+1][i] - 
                  2.0*u[k][j][i][4]*rho_i[k][j][i] +
                    u[k][j-1][i][4]*rho_i[k][j-1][i]) -
          ty2 * ((c1*u[k][j+1][i][4] - c2*square[k][j+1][i]) * vp1 -
                 (c1*u[k][j-1][i][4] - c2*square[k][j-1][i]) * vm1);
      }
    }

    //---------------------------------------------------------------------
    // add fourth order eta-direction dissipation         
    //---------------------------------------------------------------------
    if (j == 1)
        if (i >= 1 && i <= nx2) {
#pragma unroll
          for (m = 0; m < 5; m++) {
            rhs[k][j][i][m] = rhs[k][j][i][m]- dssp * 
              ( 5.0*u[k][j][i][m] - 4.0*u[k][j+1][i][m] + u[k][j+2][i][m]);
          }
        }

    if (j == 2)
        if (i >= 1 && i <= nx2) {
#pragma unroll
          for (m = 0; m < 5; m++) {
            rhs[k][j][i][m] = rhs[k][j][i][m] - dssp * 
              (-4.0*u[k][j-1][i][m] + 6.0*u[k][j][i][m] -
                4.0*u[k][j+1][i][m] + u[k][j+2][i][m]);
          }
        }

    if (j >= 3 && j <= ny2-2) {
      if (i >= 1 && i <= nx2) {
#pragma unroll
        for (m = 0; m < 5; m++) {
          rhs[k][j][i][m] = rhs[k][j][i][m] - dssp * 
            ( u[k][j-2][i][m] - 4.0*u[k][j-1][i][m] + 
            6.0*u[k][j][i][m] - 4.0*u[k][j+1][i][m] + 
              u[k][j+2][i][m] );
        }
      }
    }

    if (j == ny2-1)
        if (i >= 1 && i <= nx2) {
#pragma unroll
          for (m = 0; m < 5; m++) {
            rhs[k][j][i][m] = rhs[k][j][i][m] - dssp *
              ( u[k][j-2][i][m] - 4.0*u[k][j-1][i][m] + 
              6.0*u[k][j][i][m] - 4.0*u[k][j+1][i][m] );
          }
        }

    if (j == ny2)
        if (i >= 1 && i <= nx2) {
#pragma unroll
          for (m = 0; m < 5; m++) {
            rhs[k][j][i][m] = rhs[k][j][i][m] - dssp *
              ( u[k][j-2][i][m] - 4.0*u[k][j-1][i][m] + 5.0*u[k][j][i][m] );
          }
        }
  }
}

__global__ void compute_rhs_zeta(
    dim3 gridOffset,
    int nx2, int ny2, int nz2,
    double (*u      )/*[KMAX]*/[JMAXP+1][IMAXP+1][5],
    double (*us     )/*[KMAX]*/[JMAXP+1][IMAXP+1],
    double (*vs     )/*[KMAX]*/[JMAXP+1][IMAXP+1],
    double (*ws     )/*[KMAX]*/[JMAXP+1][IMAXP+1],
    double (*qs     )/*[KMAX]*/[JMAXP+1][IMAXP+1],
    double (*rho_i  )/*[KMAX]*/[JMAXP+1][IMAXP+1],
    double (*square )/*[KMAX]*/[JMAXP+1][IMAXP+1],
    double (*rhs    )/*[KMAX]*/[JMAXP+1][IMAXP+1][5],
    double dz1tz1, double dz2tz1, double dz3tz1, double dz4tz1, double dz5tz1, double tz2,
    double zzcon2, double zzcon3, double zzcon4, double zzcon5
) {
  int i = blockDim.x * blockIdx.x + threadIdx.x + gridOffset.x;
  int j = blockDim.y * blockIdx.y + threadIdx.y + gridOffset.y;
  int k = blockDim.z * blockIdx.z + threadIdx.z + gridOffset.z;

  int m;
  double wijk, wp1, wm1;
  if (k >= 1 && k <= nz2) {
    if (j >= 1 && j <= ny2) {
      if (i >= 1 && i <= nx2) {
        wijk = ws[k][j][i];
        wp1  = ws[k+1][j][i];
        wm1  = ws[k-1][j][i];

        rhs[k][j][i][0] = rhs[k][j][i][0] + dz1tz1 * 
          (u[k+1][j][i][0] - 2.0*u[k][j][i][0] + u[k-1][j][i][0]) -
          tz2 * (u[k+1][j][i][3] - u[k-1][j][i][3]);

        rhs[k][j][i][1] = rhs[k][j][i][1] + dz2tz1 * 
          (u[k+1][j][i][1] - 2.0*u[k][j][i][1] + u[k-1][j][i][1]) +
          zzcon2 * (us[k+1][j][i] - 2.0*us[k][j][i] + us[k-1][j][i]) -
          tz2 * (u[k+1][j][i][1]*wp1 - u[k-1][j][i][1]*wm1);

        rhs[k][j][i][2] = rhs[k][j][i][2] + dz3tz1 * 
          (u[k+1][j][i][2] - 2.0*u[k][j][i][2] + u[k-1][j][i][2]) +
          zzcon2 * (vs[k+1][j][i] - 2.0*vs[k][j][i] + vs[k-1][j][i]) -
          tz2 * (u[k+1][j][i][2]*wp1 - u[k-1][j][i][2]*wm1);

        rhs[k][j][i][3] = rhs[k][j][i][3] + dz4tz1 * 
          (u[k+1][j][i][3] - 2.0*u[k][j][i][3] + u[k-1][j][i][3]) +
          zzcon2*con43 * (wp1 - 2.0*wijk + wm1) -
          tz2 * (u[k+1][j][i][3]*wp1 - u[k-1][j][i][3]*wm1 +
                (u[k+1][j][i][4] - square[k+1][j][i] - 
                 u[k-1][j][i][4] + square[k-1][j][i]) * c2);

        rhs[k][j][i][4] = rhs[k][j][i][4] + dz5tz1 * 
          (u[k+1][j][i][4] - 2.0*u[k][j][i][4] + u[k-1][j][i][4]) +
          zzcon3 * (qs[k+1][j][i] - 2.0*qs[k][j][i] + qs[k-1][j][i]) +
          zzcon4 * (wp1*wp1 - 2.0*wijk*wijk + wm1*wm1) +
          zzcon5 * (u[k+1][j][i][4]*rho_i[k+1][j][i] - 
                  2.0*u[k][j][i][4]*rho_i[k][j][i] +
                    u[k-1][j][i][4]*rho_i[k-1][j][i]) -
          tz2 * ((c1*u[k+1][j][i][4] - c2*square[k+1][j][i])*wp1 -
                 (c1*u[k-1][j][i][4] - c2*square[k-1][j][i])*wm1);
      }
    }
  }

  //---------------------------------------------------------------------
  // add fourth order zeta-direction dissipation                
  //---------------------------------------------------------------------
  if (k == 1)
      if (j >= 1 && j <= ny2) {
        if (i >= 1 && i <= nx2) {
#pragma unroll
          for (m = 0; m < 5; m++) {
            rhs[k][j][i][m] = rhs[k][j][i][m]- dssp * 
              (5.0*u[k][j][i][m] - 4.0*u[k+1][j][i][m] + u[k+2][j][i][m]);
          }
        }
      }

  if (k == 2)
      if (j >= 1 && j <= ny2) {
        if (i >= 1 && i <= nx2) {
#pragma unroll
          for (m = 0; m < 5; m++) {
            rhs[k][j][i][m] = rhs[k][j][i][m] - dssp * 
              (-4.0*u[k-1][j][i][m] + 6.0*u[k][j][i][m] -
                4.0*u[k+1][j][i][m] + u[k+2][j][i][m]);
          }
        }
      }

  if (k >= 3 && k <= nz2-2) {
    if (j >= 1 && j <= ny2) {
      if (i >= 1 && i <= nx2) {
#pragma unroll
        for (m = 0; m < 5; m++) {
          rhs[k][j][i][m] = rhs[k][j][i][m] - dssp * 
            ( u[k-2][j][i][m] - 4.0*u[k-1][j][i][m] + 
            6.0*u[k][j][i][m] - 4.0*u[k+1][j][i][m] + 
              u[k+2][j][i][m] );
        }
      }
    }
  }

  if (k == nz2-1)
      if (j >= 1 && j <= ny2) {
        if (i >= 1 && i <= nx2) {
#pragma unroll
          for (m = 0; m < 5; m++) {
            rhs[k][j][i][m] = rhs[k][j][i][m] - dssp *
              ( u[k-2][j][i][m] - 4.0*u[k-1][j][i][m] + 
              6.0*u[k][j][i][m] - 4.0*u[k+1][j][i][m] );
          }
        }
      }

  if (k == nz2)
      if (j >= 1 && j <= ny2) {
        if (i >= 1 && i <= nx2) {
#pragma unroll
          for (m = 0; m < 5; m++) {
            rhs[k][j][i][m] = rhs[k][j][i][m] - dssp *
              ( u[k-2][j][i][m] - 4.0*u[k-1][j][i][m] + 5.0*u[k][j][i][m] );
          }
        }
      }
}

__global__ void compute_rhs_tail(
    dim3 gridOffset,
    int nx2, int ny2, int nz2,
    double (*u      )/*[KMAX]*/[JMAXP+1][IMAXP+1][5],
    double (*us     )/*[KMAX]*/[JMAXP+1][IMAXP+1],
    double (*vs     )/*[KMAX]*/[JMAXP+1][IMAXP+1],
    double (*ws     )/*[KMAX]*/[JMAXP+1][IMAXP+1],
    double (*qs     )/*[KMAX]*/[JMAXP+1][IMAXP+1],
    double (*rho_i  )/*[KMAX]*/[JMAXP+1][IMAXP+1],
    double (*square )/*[KMAX]*/[JMAXP+1][IMAXP+1],
    double (*rhs    )/*[KMAX]*/[JMAXP+1][IMAXP+1][5],
    double dt
) {
  int i = blockDim.x * blockIdx.x + threadIdx.x + gridOffset.x;
  int j = blockDim.y * blockIdx.y + threadIdx.y + gridOffset.y;
  int k = blockDim.z * blockIdx.z + threadIdx.z + gridOffset.z;

  int m;
  if (k >= 1 && k <= nz2) {
    if (j >= 1 && j <= ny2) {
      if (i >= 1 && i <= nx2) {
#pragma unroll
        for (m = 0; m < 5; m++) {
          rhs[k][j][i][m] = rhs[k][j][i][m] * dt;
        }
      }
    }
  }
}

void compute_rhs()
{
  if (timeron) timer_start(t_rhs);

  compute_rhs_intro <<< gridDim_, blockDim_ >>> (
    gridOffset, dev_grid_points, dev_u, dev_us, dev_vs, dev_ws, dev_qs, dev_rho_i, dev_speed, dev_square, dev_rhs, dev_forcing 
  );

  //---------------------------------------------------------------------
  // compute xi-direction fluxes 
  //---------------------------------------------------------------------
  if (timeron) timer_start(t_rhsx);
  compute_rhs_xi <<< gridDim_, blockDim_ >>> (
    gridOffset, nx2, ny2, nz2, dev_u, dev_us, dev_vs, dev_ws, dev_qs, dev_rho_i, dev_square, dev_rhs, dx1tx1, dx2tx1, dx3tx1, dx4tx1, dx5tx1, tx2, xxcon2, xxcon3, xxcon4, xxcon5
  );
  if (timeron) timer_stop(t_rhsx);

  //---------------------------------------------------------------------
  // compute eta-direction fluxes 
  //---------------------------------------------------------------------
  if (timeron) timer_start(t_rhsy);
  compute_rhs_eta <<< gridDim_, blockDim_ >>> (
    gridOffset, nx2, ny2, nz2, dev_u, dev_us, dev_vs, dev_ws, dev_qs, dev_rho_i, dev_square, dev_rhs, dy1ty1, dy2ty1, dy3ty1, dy4ty1, dy5ty1, ty2, yycon2, yycon3, yycon4, yycon5
  );
  if (timeron) timer_stop(t_rhsy);

  //---------------------------------------------------------------------
  // compute zeta-direction fluxes 
  //---------------------------------------------------------------------
  if (timeron) timer_start(t_rhsz);
  compute_rhs_zeta <<< gridDim_, blockDim_ >>> (
    gridOffset, nx2, ny2, nz2, dev_u, dev_us, dev_vs, dev_ws, dev_qs, dev_rho_i, dev_square, dev_rhs, dz1tz1, dz2tz1, dz3tz1, dz4tz1, dz5tz1, tz2, zzcon2, zzcon3, zzcon4, zzcon5
  );
  if (timeron) timer_stop(t_rhsz);

  compute_rhs_tail <<< gridDim_, blockDim_ >>> (
    gridOffset, nx2, ny2, nz2, dev_u, dev_us, dev_vs, dev_ws, dev_qs, dev_rho_i, dev_square, dev_rhs, dt
  );

  if (timeron) timer_stop(t_rhs);
  assert(hipSuccess == hipDeviceSynchronize());
}
