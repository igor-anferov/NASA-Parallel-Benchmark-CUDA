#include "hip/hip_runtime.h"
//-------------------------------------------------------------------------//
//                                                                         //
//  This benchmark is a serial C version of the NPB SP code. This C        //
//  version is developed by the Center for Manycore Programming at Seoul   //
//  National University and derived from the serial Fortran versions in    //
//  "NPB3.3-SER" developed by NAS.                                         //
//                                                                         //
//  Permission to use, copy, distribute and modify this software for any   //
//  purpose with or without fee is hereby granted. This software is        //
//  provided "as is" without express or implied warranty.                  //
//                                                                         //
//  Information on NPB 3.3, including the technical report, the original   //
//  specifications, source code, results and information on how to submit  //
//  new results, is available at:                                          //
//                                                                         //
//           http://www.nas.nasa.gov/Software/NPB/                         //
//                                                                         //
//  Send comments or suggestions for this C version to cmp@aces.snu.ac.kr  //
//                                                                         //
//          Center for Manycore Programming                                //
//          School of Computer Science and Engineering                     //
//          Seoul National University                                      //
//          Seoul 151-744, Korea                                           //
//                                                                         //
//          E-mail:  cmp@aces.snu.ac.kr                                    //
//                                                                         //
//-------------------------------------------------------------------------//

//-------------------------------------------------------------------------//
// Authors: Sangmin Seo, Jungwon Kim, Jun Lee, Jeongho Nah, Gangwon Jo,    //
//          and Jaejin Lee                                                 //
//-------------------------------------------------------------------------//

#include "header.h"
#include <assert.h>
#include "exact_solution.cu"

__global__ void initialize_kernel(
    dim3 gridOffset,
    int* grid_points,
    double (*u)/*[KMAX]*/[JMAXP+1][IMAXP+1][5],
    double dnxm1, double dnym1, double dnzm1
) {
  int i = blockDim.x * blockIdx.x + threadIdx.x + gridOffset.x;
  int j = blockDim.y * blockIdx.y + threadIdx.y + gridOffset.y;
  int k = blockDim.z * blockIdx.z + threadIdx.z + gridOffset.z;

  int m, ix, iy, iz;
  double xi, eta, zeta, Pface[2][3][5], Pxi, Peta, Pzeta, temp[5];

  //---------------------------------------------------------------------
  //  Later (in compute_rhs) we compute 1/u for every element. A few of 
  //  the corner elements are not used, but it convenient (and faster) 
  //  to compute the whole thing with a simple loop. Make sure those 
  //  values are nonzero by initializing the whole thing here. 
  //---------------------------------------------------------------------
  if (k >= 0 && k <= grid_points[2]-1) {
    if (j >= 0 && j <= grid_points[1]-1) {
      if (i >= 0 && i <= grid_points[0]-1) {
        u[k][j][i][0] = 1.0;
        u[k][j][i][1] = 0.0;
        u[k][j][i][2] = 0.0;
        u[k][j][i][3] = 0.0;
        u[k][j][i][4] = 1.0;
      }
    }
  }

  //---------------------------------------------------------------------
  // first store the "interpolated" values everywhere on the grid    
  //---------------------------------------------------------------------
  if (k >= 0 && k <= grid_points[2]-1) {
    zeta = (double)k * dnzm1;
    if (j >= 0 && j <= grid_points[1]-1) {
      eta = (double)j * dnym1;
      if (i >= 0 && i <= grid_points[0]-1) {
        xi = (double)i * dnxm1;

        for (ix = 0; ix < 2; ix++) {
          Pxi = (double)ix;
          exact_solution(Pxi, eta, zeta, &Pface[ix][0][0]);
        }

        for (iy = 0; iy < 2; iy++) {
          Peta = (double)iy;
          exact_solution(xi, Peta, zeta, &Pface[iy][1][0]);
        }

        for (iz = 0; iz < 2; iz++) {
          Pzeta = (double)iz;
          exact_solution(xi, eta, Pzeta, &Pface[iz][2][0]);
        }

        for (m = 0; m < 5; m++) {
          Pxi   = xi   * Pface[1][0][m] + (1.0-xi)   * Pface[0][0][m];
          Peta  = eta  * Pface[1][1][m] + (1.0-eta)  * Pface[0][1][m];
          Pzeta = zeta * Pface[1][2][m] + (1.0-zeta) * Pface[0][2][m];

          u[k][j][i][m] = Pxi + Peta + Pzeta - 
                          Pxi*Peta - Pxi*Pzeta - Peta*Pzeta + 
                          Pxi*Peta*Pzeta;
        }
      }
    }
  }


  //---------------------------------------------------------------------
  // now store the exact values on the boundaries        
  //---------------------------------------------------------------------

  //---------------------------------------------------------------------
  // west face                                                  
  //---------------------------------------------------------------------
  xi = 0.0;
  if (i == 0)
      if (k >= 0 && k <= grid_points[2]-1) {
        zeta = (double)k * dnzm1;
        if (j >= 0 && j <= grid_points[1]-1) {
          eta = (double)j * dnym1;
          exact_solution(xi, eta, zeta, temp);
          for (m = 0; m < 5; m++) {
            u[k][j][i][m] = temp[m];
          }
        }
      }

  //---------------------------------------------------------------------
  // east face                                                      
  //---------------------------------------------------------------------
  xi = 1.0;
  if (i == grid_points[0]-1)
      if (k >= 0 && k <= grid_points[2]-1) {
        zeta = (double)k * dnzm1;
        if (j >= 0 && j <= grid_points[1]-1) {
          eta = (double)j * dnym1;
          exact_solution(xi, eta, zeta, temp);
          for (m = 0; m < 5; m++) {
            u[k][j][i][m] = temp[m];
          }
        }
      }

  //---------------------------------------------------------------------
  // south face                                                 
  //---------------------------------------------------------------------
  eta = 0.0;
  if (j == 0)
      if (k >= 0 && k <= grid_points[2]-1) {
        zeta = (double)k * dnzm1;
        if (i >= 0 && i <= grid_points[0]-1) {
          xi = (double)i * dnxm1;
          exact_solution(xi, eta, zeta, temp);
          for (m = 0; m < 5; m++) {
            u[k][j][i][m] = temp[m];
          }
        }
      }

  //---------------------------------------------------------------------
  // north face                                    
  //---------------------------------------------------------------------
  eta = 1.0;
  if (j == grid_points[1]-1)
      if (k >= 0 && k <= grid_points[2]-1) {
        zeta = (double)k * dnzm1;
        if (i >= 0 && i <= grid_points[0]-1) {
          xi = (double)i * dnxm1;
          exact_solution(xi, eta, zeta, temp);
          for (m = 0; m < 5; m++) {
            u[k][j][i][m] = temp[m];
          }
        }
      }

  //---------------------------------------------------------------------
  // bottom face                                       
  //---------------------------------------------------------------------
  zeta = 0.0;
  if (k == 0)
      if (j >= 0 && j <= grid_points[1]-1) {
        eta = (double)j * dnym1;
        if (i >= 0 && i <= grid_points[0]-1) {
          xi = (double)i * dnxm1;
          exact_solution(xi, eta, zeta, temp);
          for (m = 0; m < 5; m++) {
            u[k][j][i][m] = temp[m];
          }
        }
      }

  //---------------------------------------------------------------------
  // top face     
  //---------------------------------------------------------------------
  zeta = 1.0;
  if (k == grid_points[2]-1)
      if (j >= 0 && j <= grid_points[1]-1) {
        eta = (double)j * dnym1;
        if (i >= 0 && i <= grid_points[0]-1) {
          xi = (double)i * dnxm1;
          exact_solution(xi, eta, zeta, temp);
          for (m = 0; m < 5; m++) {
            u[k][j][i][m] = temp[m];
          }
        }
      }
}

//---------------------------------------------------------------------
// This subroutine initializes the field variable u using 
// tri-linear transfinite interpolation of the boundary values     
//---------------------------------------------------------------------
void initialize()
{
  if (timeron) timer_start(t_init);
  initialize_kernel <<< gridDim_, blockDim_ >>> (gridOffset, dev_grid_points, dev_u, dnxm1, dnym1, dnzm1);
  CHK_CUDA_OK(hipDeviceSynchronize());
  if (timeron) timer_stop(t_init);
}
