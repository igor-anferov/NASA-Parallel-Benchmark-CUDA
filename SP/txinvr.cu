#include "hip/hip_runtime.h"
//-------------------------------------------------------------------------//
//                                                                         //
//  This benchmark is a serial C version of the NPB SP code. This C        //
//  version is developed by the Center for Manycore Programming at Seoul   //
//  National University and derived from the serial Fortran versions in    //
//  "NPB3.3-SER" developed by NAS.                                         //
//                                                                         //
//  Permission to use, copy, distribute and modify this software for any   //
//  purpose with or without fee is hereby granted. This software is        //
//  provided "as is" without express or implied warranty.                  //
//                                                                         //
//  Information on NPB 3.3, including the technical report, the original   //
//  specifications, source code, results and information on how to submit  //
//  new results, is available at:                                          //
//                                                                         //
//           http://www.nas.nasa.gov/Software/NPB/                         //
//                                                                         //
//  Send comments or suggestions for this C version to cmp@aces.snu.ac.kr  //
//                                                                         //
//          Center for Manycore Programming                                //
//          School of Computer Science and Engineering                     //
//          Seoul National University                                      //
//          Seoul 151-744, Korea                                           //
//                                                                         //
//          E-mail:  cmp@aces.snu.ac.kr                                    //
//                                                                         //
//-------------------------------------------------------------------------//

//-------------------------------------------------------------------------//
// Authors: Sangmin Seo, Jungwon Kim, Jun Lee, Jeongho Nah, Gangwon Jo,    //
//          and Jaejin Lee                                                 //
//-------------------------------------------------------------------------//

#include <assert.h>
#include "header.h"

//---------------------------------------------------------------------
// block-diagonal matrix-vector multiplication                  
//---------------------------------------------------------------------
__global__ void txinvr_kernel(
    dim3 gridOffset,
    int nx2, int ny2, int nz2,
    double (*us     )/*[KMAX]*/[JMAXP+1][IMAXP+1],
    double (*vs     )/*[KMAX]*/[JMAXP+1][IMAXP+1],
    double (*ws     )/*[KMAX]*/[JMAXP+1][IMAXP+1],
    double (*qs     )/*[KMAX]*/[JMAXP+1][IMAXP+1],
    double (*rho_i  )/*[KMAX]*/[JMAXP+1][IMAXP+1],
    double (*speed  )/*[KMAX]*/[JMAXP+1][IMAXP+1],
    double (*rhs    )/*[KMAX]*/[JMAXP+1][IMAXP+1][5]
) {
  int i = blockDim.x * blockIdx.x + threadIdx.x + gridOffset.x;
  int j = blockDim.y * blockIdx.y + threadIdx.y + gridOffset.y;
  int k = blockDim.z * blockIdx.z + threadIdx.z + gridOffset.z;

  double t1, t2, t3, ac, ru1, uu, vv, ww, r1, r2, r3, r4, r5, ac2inv;

  if (k >= 1 && k <= nz2) {
    if (j >= 1 && j <= ny2) {
      if (i >= 1 && i <= nx2) {
        ru1 = rho_i[k][j][i];
        uu = us[k][j][i];
        vv = vs[k][j][i];
        ww = ws[k][j][i];
        ac = speed[k][j][i];
        ac2inv = ac*ac;

        r1 = rhs[k][j][i][0];
        r2 = rhs[k][j][i][1];
        r3 = rhs[k][j][i][2];
        r4 = rhs[k][j][i][3];
        r5 = rhs[k][j][i][4];

        t1 = c2 / ac2inv * ( qs[k][j][i]*r1 - uu*r2  - vv*r3 - ww*r4 + r5 );
        t2 = bt * ru1 * ( uu * r1 - r2 );
        t3 = ( bt * ru1 * ac ) * t1;

        rhs[k][j][i][0] = r1 - t1;
        rhs[k][j][i][1] = - ru1 * ( ww*r1 - r4 );
        rhs[k][j][i][2] =   ru1 * ( vv*r1 - r3 );
        rhs[k][j][i][3] = - t2 + t3;
        rhs[k][j][i][4] =   t2 + t3;
      }
    }
  }
}

void txinvr()
{
  if (timeron) timer_start(t_txinvr);
  txinvr_kernel <<< gridDim_, blockDim_ >>> (
    gridOffset, nx2, ny2, nz2, dev_us, dev_vs, dev_ws, dev_qs, dev_rho_i, dev_speed, dev_rhs
  );
  CHK_CUDA_OK(hipDeviceSynchronize());
  if (timeron) timer_stop(t_txinvr);
}

